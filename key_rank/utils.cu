#include "hip/hip_runtime.h"
/*
 RDS: FPGA Routing Delay Sensors for Effective Remote Power Analysis Attacks
 Copyright 2023, School of Computer and Communication Sciences, EPFL.

 All rights reserved. Use of this source code is governed by a
 BSD-style license that can be found in the LICENSE.md file. 
 */

#include "utils.cuh"

void print_help() {
  printf("HELP\n");
  printf("\n==================================================\n");
  printf("CPA Key Rank Estimation Attack\n");
  printf("\n==================================================\n");
  printf("\nShort summary:\n");
  printf("\t- This program takes the power consumption traces, ciphertexts, and the last round key and computes the log2 key rank estimation metric using CPA.\n");
  printf("\t- The ouput of this program are the upper and lower bounds of the log2(key rank) metric, in a .csv file.\n");
  printf("\n==================================================\n");
  printf("\nProgram arguments:\n");
  printf("\t-h:              print help.\n");
  printf("\t-k <hexvalue>:   last round key.\n");
  printf("\t-t <file-path>:  path to trace file.\n");
  printf("\t-c <file-path>:  path to ciphertext file.\n");
  printf("\t-nt <number>:    number of encryptions (traces).\n");
  printf("\t-ns <number>:    number of samples per trace (trace lenght).\n");
  printf("\t-ss <number>:    step size for the attack.\n");
  printf("\t-o <dir-path>:   output directory.\n");
  printf("\n\n\n");

  return;
}

int parse_args(int argc, char* argv[], config_t* config) {

  int used_arguments = 0;

  if(argc == 1) {
    print_help();
    fprintf(stderr, "No arguments passed!\n");
    return EXIT_FAILURE;
  }

  if(argv == NULL) {
    fprintf(stderr, "Passed NULL argument string to parse_args\n");
    return EXIT_FAILURE;
  }

  for(int i = 1; i < argc; i++) {
    if(argv[i][1] == 'h') {
      print_help();
      exit(1);
    } else if(argv[i][1] == 'k'){
      i++;
      const char *src = argv[i];
      int buffer[16];
      int *dst = buffer;
      int *end = buffer + sizeof(buffer);
      unsigned int u;
      int counter = 0;
      while (dst < end && sscanf(src, "%2x", &u) == 1)
      {
          *dst++ = u;
          src += 2;
          counter++;
      }
      if((counter != 16) || *src != '\0'){
        printf("Given key does not have size 16. Key size must be 16 bytes.\n");
        return EXIT_FAILURE;
      }
      memcpy(config->key, buffer, sizeof(buffer));
      used_arguments++;
    } else if(argv[i][1] == 't') {
      i++;
      memcpy(config->trace_path, argv[i], strlen(argv[i]));
      config->trace_path[strlen(argv[i])] = '\0';
      used_arguments++;
    } else if(argv[i][1] == 'c') {
      i++;
      memcpy(config->ciphertext_path, argv[i], strlen(argv[i]));
      config->ciphertext_path[strlen(argv[i])] = '\0';
      used_arguments++;
    } else if(argv[i][1] == 'n' && argv[i][2] == 't') {
      i++;
      config->n_traces = atoi(argv[i]);
      used_arguments++;
    } else if(argv[i][1] == 'n' && argv[i][2] == 's') {
      i++;
      config->n_samples = atoi(argv[i]);
      used_arguments++;
    } else if(argv[i][1] == 's' && argv[i][2] == 's') {
      i++;
      config->step_size = atoi(argv[i]);
      used_arguments++;
    } else if(argv[i][1] == 'o') {
      i++;
      memcpy(config->dump_path, argv[i], strlen(argv[i]));
      config->dump_path[strlen(argv[i])] = '\0';
      used_arguments++;
    }else {
      printf("Unknown argument: -%c\n\n", argv[i][1]);
      print_help();
      return EXIT_FAILURE;
    }
  }

  if(used_arguments != 7){
    printf("Not enough arguments used. All arguments except help need to be specified!\n");
    print_help();
    return EXIT_FAILURE;
  } else {
    return EXIT_SUCCESS;
  }

}

int init_config(config_t* config){

  if (config == NULL)
    return EXIT_FAILURE;


  unsigned char key[16] = {0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00};
  memcpy(config->key, key, sizeof(config->key));
  config->dump_path[0] = '.'; 
  config->dump_path[1] = '\0'; 
  config->trace_path[0] = '.'; 
  config->trace_path[1] = '\0'; 
  config->ciphertext_path[0] = '.'; 
  config->ciphertext_path[1] = '\0'; 
  config->n_traces     = 100; 
  config->n_samples    = 128; 
  config->step_size    = 10; 
  return EXIT_SUCCESS;

}

int print_config(config_t* config){

  if (config == NULL)
    return EXIT_FAILURE;
  
  printf("\nProgram configuration:\n");
  printf("\t- key: 0x%x", config->key[0]);
  for(int j=1;j<16; j++)
    printf("%x", config->key[j]);
  printf("\n");
  printf("\t- trace file path: %s\n\n", config->trace_path);
  printf("\t- ciphertext file path: %s\n\n", config->ciphertext_path);
  printf("\t- number of traces: %d\n", config->n_traces);
  printf("\t- number of trace samples: %d\n", config->n_samples);
  printf("\t- step size for attack: %d\n", config->step_size);
  printf("\t- output path: %s\n\n", config->trace_path);

  return EXIT_SUCCESS;

}



