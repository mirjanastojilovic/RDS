#include "hip/hip_runtime.h"
/*
 RDS: FPGA Routing Delay Sensors for Effective Remote Power Analysis Attacks
 Copyright 2023, School of Computer and Communication Sciences, EPFL.

 All rights reserved. Use of this source code is governed by a
 BSD-style license that can be found in the LICENSE.md file. 
*/

/*
This source file is based on the artifact of VITI: A Tiny Self-Calibrating Sensor for Power-Variation Measurement in FPGAs
Which itself was modified from the following repo https://github.com/hasindu2008/PowerAnalysis/tree/master/4.analysis/cuda repository authored by Hasindu Gamaarachchi, Harsha Ganegoda and Roshan Ragel.
Please give due credit to the original authors by also citing their work:
* https://doi.org/10.46586/tches.v2022.i1.657-678 
* https://doi.org/10.1109/ICIINFS.2015.7399063
*/

#include "data.cuh"
#include "utils.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <stddef.h>


// GPU index
#define GPUIDXINT 0
#define GPUIDXSTR "0"

// Single run or multi run (comment the definition to switch to single run)
#define MULTIRUN

// length of the key
#define KEYBYTES 16

// there are 2^8 = 256 possibilities for each byte
#define KEYS 256

#ifdef MULTIRUN
#define ROUNDS_PER_STEP 1 // Number of CPA executions with a given number of power traces - For random selection of traces
#define MULTIRUN_SUMMARY
#endif // MULTIRUN

__device__ byte hamming_weight(byte M, byte R);
__device__ byte hamming(unsigned int *cipherText, unsigned int sample, unsigned int n, unsigned int key);
__global__ void max_correlation_kernel(double *correlation, double *waveStat, double *waveStat2, double *hammingStat, unsigned int samplesToProcess, int WAVELENGTH);
__global__ void wave_stat_kernel(double *waveData, double *waveStat, double *waveStat2, byte *hammingArray, byte *hammingArray2, unsigned int samplesToProcess, int WAVELENGTH);
__global__ void hamming_kernel(unsigned int *cipherText, byte *hammingArray,byte *hammingArray2, double *hammingStat, unsigned int samplesToProcess);

#ifdef MULTIRUN_SUMMARY
void cpa_single(int argc, char *trace_path, unsigned int *cipherTextRead, unsigned int samplesToProcess, int total, int ROUNDKEY[KEYBYTES], int WAVELENGTH, int CHUNK, char output_path[1000], unsigned int *keyByteIndex);
#endif // MULTIRUN_SUMMARY
#ifndef MULTIRUN_SUMMARY
void cpa_single(int argc, char *trace_path, unsigned int *cipherTextRead, unsigned int samplesToProcess, int total, int ROUNDKEY[KEYBYTES], int WAVELENGTH, int CHUNK, char output_path[1000]);
#endif // !MULTIRUN_SUMMARY
void randomize_selection(unsigned int *selection, unsigned int samplesToProcess);
void log_correlations_each_iteration(int iteration, double *correlation, unsigned int samplesToProcess, char output_path[1000]);
void log_maxCorrelation(double *maxCorrelation, unsigned int samplesToProcess, unsigned int file_index, char output_path[1000]);
void log_correlation_known_key_csv(double *maxCorrelation, int ROUNDKEY[KEYBYTES], char output_path[1000]);
void sort_correlations(double finalCorrelations[KEYS][KEYBYTES], int positions[KEYS][KEYBYTES], double *maxCorrelation);
void log_highest_correlation_csv(double finalCorrelations[KEYS][KEYBYTES], char output_path[1000]);
void log_top_k_correlations(double finalCorrelations[KEYS][KEYBYTES], int positions[KEYS][KEYBYTES], int k, char output_path[1000]);
void print_top_k_correlations(double finalCorrelations[KEYS][KEYBYTES], int positions[KEYS][KEYBYTES], int k);
void isMemoryFull(unsigned int *ptr);
//functions for multiple CPA attacks
void log_correct_keybyte_count_csv(int positions[KEYS][KEYBYTES], int ROUNDKEY[KEYBYTES], char output_path[1000]);
void log_misc_string(char *str, char output_path[1000]);
void multirun_update_summary(int positions[KEYS][KEYBYTES], unsigned int keyByteIndex[KEYBYTES], int ROUNDKEY[KEYBYTES]);
void log_keybyte_summary(int i, unsigned int keyByteIndex[KEYBYTES], char output_path[1000]);

int main(int argc, char *argv[]) {
	hipSetDevice(GPUIDXINT);

	FILE *file;
        config_t config;

        // Load program config passed by the command line arguments
        init_config(&config);
        if(parse_args(argc, argv, &config) == EXIT_FAILURE)
	  exit(EXIT_FAILURE);
        if(print_config(&config) == EXIT_FAILURE)
	  exit(EXIT_FAILURE);

        int SAMPLES_WAVE = config.n_traces; 
        int TOTAL = config.n_samples; 
        int STEPSIZE = config.step_size;
        int ROUNDKEY[16];
        memcpy(ROUNDKEY, config.key, sizeof(config.key));
        int WAVELENGTH = TOTAL;
        int UPPERBOUND = SAMPLES_WAVE;
        int LOWERBOUND = STEPSIZE;
        int CHUNK = TOTAL;
        char output_path[1000];
        memcpy(output_path, config.dump_path, sizeof(config.dump_path));

	unsigned int *cipherTextRead = (unsigned int *)malloc(sizeof(unsigned int) * SAMPLES_WAVE * KEYBYTES);

	isMemoryFull(cipherTextRead);

	//get ciphertexts
        printf("Ciph file: %s\n", config.ciphertext_path);
	file = fopen(config.ciphertext_path, "r");
	//isFileOK(file);
	for (int i = 0; i < SAMPLES_WAVE; i++) {
		for (int j = 0; j < KEYBYTES; j++) {
			fscanf(file, "%X", &cipherTextRead[(i / 1)*KEYBYTES + j]);
		}
	}
	printf("ciphertext: %X %X \n", cipherTextRead[SAMPLES_WAVE*KEYBYTES-1], cipherTextRead[1]);
	fclose(file);
	
#ifdef MULTIRUN

#ifdef MULTIRUN_SUMMARY
	unsigned int *keyByteIndex = (unsigned int *)malloc(sizeof(unsigned int) *  KEYBYTES);
#endif // MULTIRUN_SUMMARY
	
	int i = UPPERBOUND;
	while (i >= LOWERBOUND) {
#ifdef MULTIRUN_SUMMARY
		for (int n = 0; n < KEYBYTES; n++) {
			keyByteIndex[n] = 0;
		}
#endif // MULTIRUN_SUMMARY
		char str_i[10];
		sprintf(str_i, "%d", i);
		log_misc_string(str_i, output_path);
		for (int j = 0; j < ROUNDS_PER_STEP; j++) {
			log_misc_string(",", output_path);
#ifdef MULTIRUN_SUMMARY
			cpa_single(argc, config.trace_path, cipherTextRead, i, TOTAL, ROUNDKEY, WAVELENGTH, CHUNK, output_path, keyByteIndex);
#endif // MULTIRUN_SUMMARY
#ifndef MULTIRUN_SUMMARY
			cpa_single(argc, config.trace_path, cipherTextRead, i, TOTAL, ROUNDKEY, WAVELENGTH, CHUNK, output_path);
#endif // !MULTIRUN_SUMMARY
		}	
#ifdef MULTIRUN_SUMMARY
		log_keybyte_summary(i, keyByteIndex, output_path);

#endif //MULTIRUN_SUMMARY
		log_misc_string("\n", output_path);
		i = i - STEPSIZE;
	}
#endif // MULTIRUN

#ifndef MULTIRUN
	cpa_single(argc, config.trace_path, cipherTextRead, SAMPLES_WAVE, TOTAL, ROUNDKEY, WAVELENGTH, CHUNK, output_path);
#endif // !MULTIRUN

#ifdef MULTIRUN_SUMMARY
	free(keyByteIndex);
#endif //MULTIRUN_SUMMARY
	free(cipherTextRead);
	return 0;
}


#ifdef MULTIRUN_SUMMARY
void cpa_single(int argc, char *trace_path, unsigned int *cipherTextRead, unsigned int samplesToProcess, int total, int ROUNDKEY[KEYBYTES], int WAVELENGTH, int CHUNK, char output_path[1000], unsigned int *keyByteIndex) {
#endif // MULTIRUN_SUMMARY
#ifndef MULTIRUN_SUMMARY
void cpa_single(int argc, char *trace_path, unsigned int *cipherTextRead, unsigned int samplesToProcess, int total, int ROUNDKEY[KEYBYTES], int WAVELENGTH, int CHUNK, char output_path[1000]) {
#endif // !MULTIRUN_SUMMARY
	FILE *file;
	float dat;
	unsigned int i, j, k, temp;

	double *maxCorrelation = (double *)malloc(sizeof(double) * KEYS* KEYBYTES);
	isMemoryFull( (unsigned int*) maxCorrelation);
	for (i = 0; i < KEYS; i++) {
		for (j = 0; j < KEYBYTES; j++) {
			maxCorrelation[i*KEYBYTES + j] = 0;
		}
	}

	double *waveDataRead = (double *)malloc(sizeof(double) * samplesToProcess * CHUNK);
	
	isMemoryFull((unsigned int*)  waveDataRead);

	//space for correlation
	double *correlation = (double *)malloc(sizeof(double) * KEYS * KEYBYTES);
	isMemoryFull((unsigned int*)correlation);

	unsigned int *selection = (unsigned int *)malloc(sizeof(unsigned int) * samplesToProcess);

	isMemoryFull((unsigned int*)selection);

	for (i = 0; i < samplesToProcess; i++) {
		selection[i] = i;
	}
	//randomize_selection(selection, samplesToProcess);

	int numOfChunks = total / CHUNK;
	int l = 0;
	for (l = 0; l < numOfChunks; l++) {
		file = fopen(trace_path, "r");
		//isFileOK(file);
		int fileLength = strlen(trace_path);
		char extention[5];
		strncpy(extention, trace_path + fileLength - 4, 4);
		extention[4] = 0;
		if (strcmp(extention, "data") == 0) {
			fprintf(stderr, "%s\n", ".data file detected");

			for (i = 0; i < 1 * samplesToProcess; i++) {
				fseek(file, sizeof(dat) * CHUNK * l, SEEK_CUR);

				temp = 0;
				for (j = 0; j < CHUNK; j++) {
					fread((void*)(&dat), sizeof(dat), 1, file);
					waveDataRead[(i / 1) * CHUNK + j] = (double)(dat);
				if(i==samplesToProcess-1 && j==CHUNK-1){
						printf("wave data %d %f \n",i*CHUNK,  waveDataRead[i*CHUNK + j] );
					}
				}
				
				fseek(file, sizeof(dat) * (total - (CHUNK  * (l + 1))), SEEK_CUR);
			}
		}
		else {
			long int dat;
			fprintf(stderr, "%s\n", ".txt file detected");
			for (i = 0; i < samplesToProcess; i++) {
				for (j = 0; j < WAVELENGTH; j++) {
					fscanf(file, "%d", &dat);
					waveDataRead[i*CHUNK + j] = (double)dat;
					if(i==samplesToProcess-1 && j==WAVELENGTH-1){
						printf("wave data %f \n", waveDataRead[i*CHUNK + j] );
					}
				}
			}
			
		}

		fclose(file);

		unsigned int innerRounds = CHUNK / WAVELENGTH;
		if (CHUNK % WAVELENGTH != 0)
			innerRounds++;
		// main loop
		for (k = 0; k < innerRounds; k++) {
			//get wave data
			double *waveData = (double *)malloc(sizeof(double) * samplesToProcess *  WAVELENGTH);
			isMemoryFull((unsigned int*)waveData);
			unsigned int *cipherText = (unsigned int *)malloc(sizeof(unsigned int) * samplesToProcess * KEYBYTES);
			isMemoryFull(cipherText);

			fprintf(stderr, "%s %d %d %d \n", "Calculating", l, k, innerRounds);

			for (i = 0; i < samplesToProcess; i++) {
				if(memcpy(&waveData[i * WAVELENGTH], &waveDataRead[selection[i] * CHUNK + k * WAVELENGTH], sizeof(double) * WAVELENGTH) == NULL){
					printf("mem cpy failed\n");
				}
				if(memcpy(&cipherText[i * KEYBYTES], &cipherTextRead[selection[i] * KEYBYTES], sizeof(unsigned int) * KEYBYTES) == NULL){
					printf("mem cpy failed\n");
				}
			}
	        free(waveDataRead);
	        free(selection);


			unsigned int *dev_cipherText;
			double *dev_correlation, *dev_waveStat, *dev_waveStat2, *dev_hammingStat, *dev_waveData;
			byte *dev_hammingArray, *dev_hammingArray2;

			if(hipMalloc((void**)&dev_waveData, 1L * samplesToProcess * WAVELENGTH * sizeof(double)) != hipSuccess){
				printf("cuda malloc failed wave data \n");
			}
			if(hipMalloc((void**)&dev_cipherText, 1L * samplesToProcess * KEYBYTES * sizeof(unsigned int)) != hipSuccess){
				printf("cuda malloc failed ciphertext\n");
			}

			if(hipMalloc((void**)&dev_hammingArray, 1L * KEYS * KEYBYTES * samplesToProcess * sizeof(byte))!= hipSuccess){
				printf("cuda malloc failed hamming array\n");
				printf("samples to process %ld \n", 1L* KEYS * KEYBYTES * samplesToProcess);
			}
			unsigned long a =  KEYS * KEYBYTES * sizeof(byte);
			double len_array = 1L * a* samplesToProcess;

			if(len_array > 4294967295){
				hipMalloc((void**)&dev_hammingArray2, a * samplesToProcess  - 4294967295 );
			}else{
				hipMalloc((void**)&dev_hammingArray2, 1 );

			}

			if(hipMalloc((void**)&dev_hammingStat, 2 * KEYS * KEYBYTES * sizeof(double)) != hipSuccess){
				printf("cuda malloc failed hammingstat");
			}

			
			if(hipMemcpy(dev_cipherText, cipherText, 1L * samplesToProcess * KEYBYTES * sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess){
				printf("cuda mem cpy failed\n");
			}
			free(cipherText);

			//find hamming model
			dim3 grid(KEYBYTES / 16, KEYS / 16);
			dim3 block(16, 16);
			hamming_kernel << <grid, block >> > (dev_cipherText, dev_hammingArray, dev_hammingArray2, dev_hammingStat, samplesToProcess);
			hipGetLastError();
			hipFree(dev_cipherText);

			//find wave stats
			if(hipMemcpy(dev_waveData, waveData, 1L * samplesToProcess * WAVELENGTH * sizeof(double), hipMemcpyHostToDevice) != hipSuccess){
				printf("cuda mem cpy failed\n");
			}
			printf("wave data %zu", 1L * samplesToProcess * WAVELENGTH * sizeof(double) - 1);
			printf("wave data %zd", 1L * samplesToProcess * WAVELENGTH * sizeof(double) - 1);
			free(waveData);

			if(hipMalloc((void**)&dev_waveStat, 2 * WAVELENGTH * sizeof(double)) != hipSuccess){
				printf("cuda malloc failed wave stat\n");
			}
			if(hipMalloc((void**)&dev_waveStat2, 1L * KEYS * KEYBYTES * WAVELENGTH * sizeof(double)) != hipSuccess){
				printf("cuda malloc failed wavestat2\n");
			}
			dim3 block3d(16, 16, 4);
			dim3 grid3d(KEYBYTES / 16, KEYS / 16, WAVELENGTH / 4);
			wave_stat_kernel << <grid3d, block3d >> > (dev_waveData, dev_waveStat, dev_waveStat2, dev_hammingArray,dev_hammingArray2, samplesToProcess, WAVELENGTH);
			hipGetLastError();
			if(hipFree(dev_waveData)!=hipSuccess){
				printf("cuda free failed\n");
			}
			if(hipFree(dev_hammingArray)!=hipSuccess){
				printf("cuda free failed\n");
			}
			if(hipFree(dev_hammingArray2)!=hipSuccess){
				printf("cuda free failed\n");
			}

			//calculate correlation coefficient
			if(hipMalloc((void**)&dev_correlation, KEYS * KEYBYTES * sizeof(double)) != hipSuccess){
				printf("cuda malloc failed correlation\n");
			}
			max_correlation_kernel << <grid, block >> > (dev_correlation, dev_waveStat, dev_waveStat2, dev_hammingStat, samplesToProcess, WAVELENGTH);
			//printf("correlation %f\n", dev_correlation[0]);
			//printf("correlation2 %f\n", dev_correlation[KEYS * KEYBYTES - 1]);

			hipGetLastError();

			//copy back to host and free
			if(hipMemcpy(correlation, dev_correlation, KEYS * KEYBYTES * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess){
				printf("cuda mem cpy failed\n");
			}
			if(hipFree(dev_correlation)!=hipSuccess){
				printf("cuda free failed\n");
			}
			if(hipFree(dev_waveStat) != hipSuccess){
				printf("cuda free failed\n");
			}
			if(hipFree(dev_waveStat2)!=hipSuccess){
				printf("cuda free failed\n");
			}
			if(hipFree(dev_hammingStat)!=hipSuccess){
				printf("cuda free failed\n");
			}
			for (i = 0; i < KEYS; i++) {
				for (j = 0; j < KEYBYTES; j++) {
					double maxValue = maxCorrelation[i * KEYBYTES + j];
					double thisIteration = correlation[i * KEYBYTES + j];
					if (maxValue < thisIteration)
						maxCorrelation[i * KEYBYTES + j] = thisIteration;
				}
			}

			//log_correlations_each_iteration(l + innerRounds * k, correlation, samplesToProcess, output_path);
		}

	}
	free(correlation);
	log_maxCorrelation(maxCorrelation, samplesToProcess, samplesToProcess, output_path);

	//log_correlation_known_key_csv(maxCorrelation, ROUNDKEY, output_path);

	double finalCorrelations[KEYS][KEYBYTES];
	int positions[KEYS][KEYBYTES];
	printf("sort\n");
	sort_correlations(finalCorrelations, positions, maxCorrelation);
	printf("sort done\n");
	free(maxCorrelation);

	//log_highest_correlation_csv(finalCorrelations, output_path);

	//log_top_k_correlations(finalCorrelations, positions, 5, output_path);

	//print_top_k_correlations(finalCorrelations, positions, 5);
	
#ifdef MULTIRUN_SUMMARY
	multirun_update_summary(positions, keyByteIndex, ROUNDKEY);
#endif // MULTIRUN_SUMMARY

#ifdef MULTIRUN
	log_correct_keybyte_count_csv(positions, ROUNDKEY, output_path);
#endif // MULTIRUN

	return;
}

__device__ byte hamming_weight(byte M, byte R) {
	byte H = M ^ R;
	// Count the number of set bits
	byte dist = 0;
	while (H) {
		dist++;
		H &= H - 1;
	}
	return dist;
}

//3rd argument n is the index of the key byte
__device__ byte hamming(unsigned int *cipherText, unsigned int sample, unsigned int n, unsigned int key) {
	byte st10 = (byte)cipherText[sample * KEYBYTES + inv_shift[n]];
	byte st9 = (byte)inv_sbox[cipherText[sample * KEYBYTES + n] ^ key];
	byte dist = hamming_weight(st9, st10);
	return dist;
}

__global__ void max_correlation_kernel(double *correlation, double *waveStat, double *waveStat2, double *hammingStat, unsigned int samplesToProcess, int WAVELENGTH) {
		int keyguess = blockDim.y * blockIdx.y + threadIdx.y;
	int keybyte = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (keybyte < KEYBYTES && keyguess < KEYS) {
		double sigmaH, sigmaH2, sigmaW = 0, sigmaW2 = 0, sigmaWH = 0;
		sigmaH = hammingStat[KEYBYTES * keyguess + keybyte];
		sigmaH2 = hammingStat[KEYS * KEYBYTES + KEYBYTES * keyguess + keybyte];
		double correlationTemp = 0;
		double correlationMax = 0;
		unsigned int j;

		for (j = 0; j < WAVELENGTH; j++) {
			sigmaWH = waveStat2[j * KEYS * KEYBYTES + keyguess * KEYBYTES + keybyte];
			sigmaW = waveStat[j];
			sigmaW2 = waveStat[WAVELENGTH + j];
			double numerator = samplesToProcess * sigmaWH - sigmaW * sigmaH;
			double denominator = sqrt(samplesToProcess * sigmaW2 - sigmaW * sigmaW) * sqrt(samplesToProcess * sigmaH2 - sigmaH * sigmaH);
			correlationTemp = fabs(numerator / denominator);

			if (correlationTemp > correlationMax) {
				correlationMax = correlationTemp;
			}
		}
		correlation[keyguess * KEYBYTES + keybyte] = correlationMax;
	}
	return;
}

__global__ void wave_stat_kernel(double *waveData, double *waveStat, double *waveStat2, byte *hammingArray, byte *hammingArray2, unsigned int samplesToProcess, int WAVELENGTH) {
	int keyguess = blockDim.y * blockIdx.y + threadIdx.y;
	int keybyte = blockDim.x * blockIdx.x + threadIdx.x;
	int wave = blockDim.z * blockIdx.z + threadIdx.z;

	if (keyguess < KEYS && keybyte < KEYBYTES && wave < WAVELENGTH) {
		unsigned int i;
		double sigmaWH = 0;
		for (i = 0; i < samplesToProcess; i++) {
			unsigned long a= KEYS * KEYBYTES;
			if((i * a + keyguess * KEYBYTES + keybyte) < 4294967295){
			sigmaWH += waveData[i * WAVELENGTH + wave] * (double)hammingArray[i * a + keyguess * KEYBYTES + keybyte];
			}else{
			sigmaWH += waveData[i * WAVELENGTH + wave] * (double)hammingArray2[(i * a + keyguess * KEYBYTES + keybyte) - 4294967295];
			}
		}
		waveStat2[wave * KEYS * KEYBYTES + keyguess * KEYBYTES + keybyte] = sigmaWH;
	}

	if (keyguess == 0 && keybyte == 0 && wave < WAVELENGTH) {
		unsigned int i;
		double sigmaW = 0, sigmaW2 = 0, W = 0;
		for (i = 0; i < samplesToProcess; i++) {
			W = waveData[i * WAVELENGTH + wave];
			sigmaW += W;
			sigmaW2 += W * W;
		}
		waveStat[wave] = sigmaW;
		waveStat[WAVELENGTH + wave] = sigmaW2;
	}
	return;
}

__global__ void hamming_kernel(unsigned int *cipherText, byte *hammingArray, byte *hammingArray2, double *hammingStat, unsigned int samplesToProcess) {
	int keyguess = blockDim.y * blockIdx.y + threadIdx.y;
	int keybyte = blockDim.x * blockIdx.x + threadIdx.x;

	if (keybyte < KEYBYTES && keyguess < KEYS) {
		double sigmaH = 0, sigmaH2 = 0;
		byte H;
		unsigned int i;
		for (i = 0; i < samplesToProcess; i++) {
			H = hamming(cipherText, i, keybyte, keyguess);
			unsigned long a = KEYS * KEYBYTES;
			if((i *a + keyguess * KEYBYTES + keybyte) < 4294967295){	
			hammingArray[i * KEYS * KEYBYTES + keyguess * KEYBYTES + keybyte] = H;
			}else{
			hammingArray2[(i * a + keyguess * KEYBYTES + keybyte) - 4294967295] = H;
			}
			sigmaH += (double)H;
			sigmaH2 += (double)H * (double)H;
		}
		hammingStat[KEYBYTES * keyguess + keybyte] = sigmaH;
		hammingStat[KEYS * KEYBYTES + KEYBYTES * keyguess + keybyte] = sigmaH2;
	}
	return;
}

void randomize_selection(unsigned int *selection, unsigned int samplesToProcess) {
	srand(time(0));
	unsigned int temp = 0;
	for (int i = 0; i < samplesToProcess; i++) {
		unsigned int swap_i = rand() % samplesToProcess;
		temp = selection[i];
		selection[i] = selection[swap_i];
		selection[swap_i] = temp;
	}
	return;
}

void log_correlations_each_iteration(int iteration, double *correlation, unsigned int samplesToProcess, char output_path[1000]) {
        char file_name[1000];
        snprintf(file_name, sizeof(char) * 1000, "%s/all_kr_" GPUIDXSTR ".txt", output_path);
	FILE *file;
	if (iteration == 0)
		file = fopen(file_name, "w");
	else
		file = fopen(file_name, "a");

	fprintf(file, "%d,  pk0,  pk1,  pk2,  pk3,  pk4,  pk5,  pk6,  pk7,  pk8,  pk9, pk10, pk11, pk12, pk13, pk14, pk15, \n", samplesToProcess);
	for (int i = 0; i < KEYS; i++) {
		fprintf(file, "0x%02X,", i);
		for (int j = 0; j < KEYBYTES; j++) {
			fprintf(file, "%.15f,", i, correlation[i * KEYBYTES + j]);
		}
		fprintf(file, "\n");
	}

	fprintf(file, "\n\n");
	fclose(file);
	return;
}

//Among the multiple iterations, the maximum correlation for each key byte and key guess
void log_maxCorrelation(double *maxCorrelation, unsigned int samplesToProcess, unsigned int file_index, char output_path[1000]) {
  char file_name[1000];
  snprintf(file_name, sizeof(char) * 1000, "%s/final_kr/%i.txt", output_path, file_index);
  
	FILE *file = fopen(file_name, "a");
	for (int i = 0; i < KEYS; i++) {
		for (int j = 0; j < KEYBYTES-1; j++) {
			fprintf(file, "%.15f,", maxCorrelation[i * KEYBYTES + j]);
		}
		fprintf(file, "%.15f\n", maxCorrelation[i * KEYBYTES + KEYBYTES - 1]);
	}
  fclose(file);
	return;
}

void log_correlation_known_key_csv(double *maxCorrelation, int ROUNDKEY[KEYBYTES], char output_path[1000]) {
	//int key[KEYBYTES] = { ROUNDKEY };
	int key[KEYBYTES]; 
        for(int i=0;i<16; i++)
          key[i] = ROUNDKEY[i];

        char file_name[1000];
        snprintf(file_name, sizeof(char) * 1000, "%s/corr_coef_key_kr_" GPUIDXSTR ".csv", output_path);
	FILE *file = fopen(file_name, "a");

	for (int i = 0; i < KEYBYTES; i++) {
		for (int j = 0; j < KEYS; j++) {
			if (key[i] == j) {
				fprintf(file, "%.15f", maxCorrelation[j * KEYBYTES + i]);
				if (i < KEYBYTES - 1)
					fprintf(file, ", ");
			}
		}
	}
	fprintf(file, "\n");
	fclose(file);
	return;
}

void sort_correlations(double finalCorrelations[KEYS][KEYBYTES], int positions[KEYS][KEYBYTES], double *maxCorrelation) {
	double n = 0;
	for (int j = 0; j < KEYBYTES; j++) {
		for (int i = 0; i < KEYS; i++) {
			finalCorrelations[i][j] = maxCorrelation[i * KEYBYTES + j];
			positions[i][j] = i;
		}
		for (int p = 0; p < 255; p++) {
			for (int i = 0; i < KEYS - p - 1; i++) {
				if (finalCorrelations[i][j] < finalCorrelations[i + 1][j]) {
					n = finalCorrelations[i][j];
					finalCorrelations[i][j] = finalCorrelations[i + 1][j];
					finalCorrelations[i + 1][j] = n;

					n = positions[i][j];
					positions[i][j] = positions[i + 1][j];
					positions[i + 1][j] = n;
				}
			}
		}
	}
	return;
}

void log_highest_correlation_csv(double finalCorrelations[KEYS][KEYBYTES], char output_path[1000]) {
        char file_name[1000];
        snprintf(file_name, sizeof(char) * 1000, "%s/corr_coef_highest_kr_" GPUIDXSTR ".csv", output_path);
	FILE *file = fopen(file_name, "a");

	for (int j = 0; j < KEYBYTES; j++) {
		fprintf(file, "%.15f", finalCorrelations[0][j]);
		if (j < KEYBYTES - 1) {
			fprintf(file, ", ");
		}
	}
	fprintf(file, "\n");
	fclose(file);
	return;
}

void log_top_k_correlations(double finalCorrelations[KEYS][KEYBYTES], int positions[KEYS][KEYBYTES], int k, char output_path[1000]) {
	FILE *file;
	char filename[1000];
	char str_k[4];
	sprintf(str_k, "%d", k);
        snprintf(filename, sizeof(char) * 1000, "%s/top_%s_keys.txt", output_path, str_k);
	file = fopen(filename, "a");

	for (int j = 0; j < KEYBYTES; j++) {
		fprintf(file, "  |%02d|\t", j);
	}
	fprintf(file, "\n");

	for (int i = 0; i < k; i++) {
		for (int j = 0; j < KEYBYTES; j++) {
			fprintf(file, "  %02x\t", positions[i][j]);
		}
		fprintf(file, "\n");
		for (int j = 0; j < KEYBYTES; j++) {
			fprintf(file, "%.15f \t", finalCorrelations[i][j]);
		}
		fprintf(file, "\n\n");
	}
	fprintf(file, "\n\n");
	fclose(file);
	return;
}

void print_top_k_correlations(double finalCorrelations[KEYS][KEYBYTES], int positions[KEYS][KEYBYTES], int k) {
	for (int j = 0; j < KEYBYTES; j++) {
		printf("  |%02d|\t", j);
	}
	printf("\n");

	for (int i = 0; i < k; i++) {
		for (int j = 0; j < KEYBYTES; j++) {
			printf("  %02x\t", positions[i][j]);
		}
		printf("\n");
		for (int j = 0; j < KEYBYTES; j++) {
			printf("%.15f \t", finalCorrelations[i][j]);
		}
		printf("\n\n");
	}
	printf("\n\n");
	return;
}

void log_correct_keybyte_count_csv(int positions[KEYS][KEYBYTES], int ROUNDKEY[KEYBYTES], char output_path[1000]) {
	//int key[KEYBYTES] = { ROUNDKEY };
	int key[KEYBYTES]; 
        for(int i=0;i<16; i++)
          key[i] = ROUNDKEY[i];

        char file_name[1000];
        snprintf(file_name, sizeof(char) * 1000, "%s/correct_keybyte_count_kr_" GPUIDXSTR ".csv", output_path);
	FILE *file = fopen(file_name, "a");
	int cnt = 0;
	for (int j = 0; j < KEYBYTES; j++) {
		if (positions[0][j] == key[j])
			cnt++;
	}
	printf("cnt %d \n", cnt);
	fprintf(file, "%d", cnt);
	fclose(file);
	return;
}

void log_misc_string(char *str, char output_path[1000]) {
        char file_name[1000];
        snprintf(file_name, sizeof(char) * 1000, "%s/correct_keybyte_count_kr_" GPUIDXSTR ".csv", output_path);
	FILE *file = fopen(file_name, "a");
	fprintf(file, "%s", str);
	fclose(file);
	return;
}

void multirun_update_summary(int positions[KEYS][KEYBYTES], unsigned int keyByteIndex[KEYBYTES], int ROUNDKEY[KEYBYTES]) {
	//int key[KEYBYTES] = { ROUNDKEY };
	int key[KEYBYTES]; 
        for(int i=0;i<16; i++)
          key[i] = ROUNDKEY[i];

	for (int j = 0; j < KEYBYTES; j++) {
		for (int i = 0; i < KEYS; i++) {
			if (positions[i][j] == key[j])
				keyByteIndex[j] = keyByteIndex[j] + i;
		}
	}
	return;
}

void log_keybyte_summary(int i, unsigned int keyByteIndex[KEYBYTES], char output_path[1000]) {
        char file_name[1000];
        snprintf(file_name, sizeof(char) * 32, "%s/summary_keybyte_kr_" GPUIDXSTR ".csv", output_path);
	FILE *file = fopen(file_name, "a");
	fprintf(file, "%d", i);
	for (int j = 0; j < KEYBYTES; j++) {
		fprintf(file, ", %d", keyByteIndex[j]);
	}
	fprintf(file, "\n");
	fclose(file);
	return;
}

void isMemoryFull(unsigned int *ptr){
	if(ptr == NULL){
		printf("----memory\n");
	}
}
